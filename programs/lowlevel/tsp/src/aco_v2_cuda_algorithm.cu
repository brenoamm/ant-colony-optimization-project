#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <fstream>
#include <stdlib.h>
#include <sstream>
#include <malloc.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>
#include <chrono>

#define CUDA_ERROR_CHECK

#include "../include/aco_v2_cuda_algorithm.cuh"

#include "Randoms.cpp"

#define PHERINIT 0.005
#define EVAPORATION 0.5
#define ALPHA 1
#define BETA 2
#define TAUMAX 2
#define BLOCK_SIZE 32
#define IROULETE 32

__device__ double d_PHERINIT;
__device__ double d_EVAPORATION;
__device__ double d_ALPHA;
__device__ double d_BETA ;
__device__ double d_TAUMAX;
__device__ int d_BLOCK_SIZE;
__device__ int d_GRAPH_SIZE;


int NBLOCKS = 0;

std::string::size_type sz;

Randoms *randoms;

// nvcc -o a.out acoCuda.cu
// ./a.out 38 50 50 3 0

using namespace std;

void readMap(double* coord, double* phero, double* dist, int n_cities, int problem){ // small

	printf(" Reading map: %d \n", problem);

	std::ifstream lerMapa;

//	    std::string dji = "/home/bamm/Documents/MUESLI/ACO/ACO-low-level/build/Debug/djibouti.txt";
//	    std::string lux = "/home/bamm/Documents/MUESLI/ACO/ACO-low-level/build/Debug/luxembourg.txt";
//	    std::string cat = "/home/bamm/Documents/MUESLI/ACO/ACO-low-level/build/Debug/catar.txt";
//	    std::string a280 = "/home/bamm/Documents/MUESLI/ACO/ACO-low-level/build/Debug/a280.txt";
//	    std::string d198 = "/home/bamm/Documents/MUESLI/ACO/ACO-low-level/build/Debug/d198.txt";
//	    std::string d1291 = "/home/bamm/Documents/MUESLI/ACO/ACO-low-level/build/Debug/d1291.txt";
//	    std::string lin318 = "/home/bamm/Documents/MUESLI/ACO/ACO-low-level/build/Debug/lin318.txt";
//	    std::string pcb442 = "/home/bamm/Documents/MUESLI/ACO/ACO-low-level/build/Debug/pcb442.txt";
//	    std::string pcb1173 = "/home/bamm/Documents/MUESLI/ACO/ACO-low-level/build/Debug/pbc1173.txt";
//	    std::string pr1002 = "/home/bamm/Documents/MUESLI/ACO/ACO-low-level/build/Debug/pr1002.txt";
//	    std::string pr2392 = "/home/bamm/Documents/MUESLI/ACO/ACO-low-level/build/Debug/pr2392.txt";
//	    std::string rat783 = "/home/bamm/Documents/MUESLI/ACO/ACO-low-level/build/Debug/rat783.txt";

		std::string dji = "/home/b/b_mene01/tsp/djibouti.txt";
		std::string lux = "/home/b/b_mene01/tsp/luxembourg.txt";
		std::string cat = "/home/b/b_mene01/tsp/catar.txt";
		std::string a280 = "/home/b/b_mene01/tsp/a280.txt";
		std::string d198 = "/home/b/b_mene01/tsp/d198.txt";
		std::string d1291 = "/home/b/b_mene01/tsp/d1291.txt";
		std::string lin318 = "/home/b/b_mene01/tsp/lin318.txt";
		std::string pcb442 = "/home/b/b_mene01/tsp/pcb442.txt";
		std::string pcb1173 = "/home/b/b_mene01/tsp/pbc1173.txt";
		std::string pr1002 = "/home/b/b_mene01/tsp/pr1002.txt";
		std::string pr2392 = "/home/b/b_mene01/tsp/pr2392.txt";
		std::string rat783 = "/home/b/b_mene01/tsp/rat783.txt";

	    switch (problem) {
			case 1:
				lerMapa.open(dji, std::ifstream::in);
				break;
			case 2:
				lerMapa.open(lux, std::ifstream::in);
				break;
			case 3:
				lerMapa.open(cat, std::ifstream::in);
				break;
			case 4:
				lerMapa.open(a280, std::ifstream::in);
				break;
			case 5:
				lerMapa.open(d198, std::ifstream::in);
				break;
			case 6:
				lerMapa.open(d1291, std::ifstream::in);
				break;
			case 7:
				lerMapa.open(lin318, std::ifstream::in);
				break;
			case 8:
				lerMapa.open(pcb442, std::ifstream::in);
				break;
			case 9:
				lerMapa.open(pcb1173, std::ifstream::in);
				break;
			case 10:
				lerMapa.open(pr1002, std::ifstream::in);
				break;
			case 11:
				lerMapa.open(pr2392, std::ifstream::in);
				break;
			case 12:
				lerMapa.open(rat783, std::ifstream::in);
				break;
		}

    if (lerMapa.is_open()) {

		double randn = 0.0;

		for(int j = 0;j<n_cities;j++){
			for(int k = 0;k<n_cities;k++){
				if(j!=k){
					randn = randoms -> Uniforme() * TAUMAX;
					phero[(j*n_cities) + k] = randn;
					phero[(k*n_cities) + j] = randn;
				}
				else{
					phero[(j*n_cities) + k] = 0;
					phero[(k*n_cities) + j] = 0;
				}
			}
		}

		int i = 0;

		double index, x, y;

		index = 0.0;
				x = 0.0;
				y = 0.0;

		while(!lerMapa.eof()){

			lerMapa >> index;
			lerMapa >> x;
			lerMapa >> y;

			coord[(i*2)] = (double)x;
			coord[(i*2) + 1] = (double)y;

			i+=1;
		}

    }    else{
    	printf(" File not opened\n");
    }
    lerMapa.close();
}


__global__ void setup_rand_kernel(hiprandState * state, unsigned long seed) {

	int id = blockIdx.x * blockDim.x + threadIdx.x;

	hiprand_init(seed, id, 0, &state[id]);
//	hiprand_init(1234, id, 0, &state[id]);

  if(id == 0){
	  d_PHERINIT = 0.005;
	  d_EVAPORATION = 0.5;
	  d_ALPHA = 1;
	  d_BETA = 2;
	  d_TAUMAX = 2;
  }

  __syncthreads();
}

__global__ void calculate_distance_kernel(double* dist, double* coord, int n_cities){

    int c_index = threadIdx.x;

    for(int j = 0 ;j<n_cities;j++){
         if(c_index!=j){
            dist[(c_index*n_cities) + j] = sqrt(pow(coord[j*2] - coord[c_index*2],2) + pow(coord[(j*2) + 1] - coord[(c_index*2) + 1],2));
            dist[(j*n_cities) + c_index] = dist[(c_index*n_cities) + j];
         }else{
        	dist[(c_index*n_cities) + j] = 0.0;
         }
    }


}

__global__ void calculate_iroulette_kernel(double* dist, double* coord, int* iroulette, int n_cities){

	int c_index = threadIdx.x;

    //Get the 32 closest nodes for each node.
    for(int i = 0 ; i < IROULETE ; i++){

    	double distance = 999999.9;
    	double c_dist = 0.0;
    	int city = -1;

		for(int j = 0 ;j<n_cities;j++){

			bool check = true;

			for(int k = 0 ; k < i ; k++){
				if(iroulette[c_index * IROULETE + k] == j){
					check = false;
				}
			}

			if(c_index!=j && check){
				c_dist = dist[(c_index*n_cities) + j];
				if(c_dist < distance){
					distance = c_dist;
					city = j;
				}
			}
		}
		iroulette[c_index * IROULETE + i] = city;
	}
}

__global__ void route_kernel(int n_cities, int* routes, double* c_phero, double* c_dist, double* d_probs, hiprandState* rand_states, double* d_eta, double* d_tau, double* d_sum){

	int ant_index = blockIdx.x;
	int dim_index = threadIdx.x;
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	int initialCity = 0;

	//set Initial City for each Ant
	if(dim_index == 0){
		routes[ant_index * n_cities] = initialCity;
	}

	//synchronize
	__syncthreads();

	//start route steps
	for (int i=0; i < n_cities-1; i++) {

		int cityi = routes[ant_index*n_cities+i];

		if(dim_index == 0){
			d_sum[ant_index] = 0.0;
		}

		//synchronize
		__syncthreads();

		d_eta[index] = 0.0;
		d_tau[index] = 0.0;

		if (cityi != dim_index && !vizited(ant_index, dim_index, routes, n_cities, i)){
			d_eta[index] = (double) pow (1 / c_dist[cityi*n_cities+dim_index], d_BETA);
			d_tau[index] = (double) pow (c_phero[(cityi*n_cities)+dim_index],   d_ALPHA);
		}

		if(dim_index == 0){
			for(int j = 0 ; j<n_cities ; j++){
				d_sum[ant_index] += d_eta[ant_index*n_cities+j] * d_tau[ant_index*n_cities+j];
			}
		}

		//synchronize
		__syncthreads();


		//calculate probability to go to city J
		int cityj = dim_index;

		if (cityi == cityj || vizited(ant_index, cityj, routes, n_cities, i)) {
			d_probs[index] = 0;
		}else{
			d_probs[index] = d_eta[index] * d_tau[index] / d_sum[ant_index];
		}

		//choose next city
		if(dim_index == 0){
			int nextCity = city(ant_index, n_cities, d_probs, rand_states);
			routes[(ant_index * n_cities) + (i + 1)] = nextCity;
		}

		//synchronize
		__syncthreads();
	}
}

__global__ void route_kernel2(int n_cities, int* routes, double* c_phero, double* c_dist, double* d_probs,  int* iroulette, hiprandState* rand_states, double* d_eta, double* d_tau,  double* d_sum){

	int ant_index = blockIdx.x;
	int dim_index = threadIdx.x;
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	int next_city = -1;

	routes[ant_index * n_cities] = 0;

	d_sum[ant_index] = 0.0;
	d_eta[index] = 0.0;
	d_tau[index] = 0.0;
	d_probs[index] = 0.0;

	//Loop to build complete route
	for (int i=0; i < n_cities-1; i++) {

		int cityi = routes[ant_index*n_cities+i];

		next_city =  iroulette[(cityi * IROULETE) + dim_index];

		if (cityi != next_city && !vizited(ant_index, next_city, routes, n_cities, i)){
			d_eta[index] = (double) pow (1 / c_dist[cityi*n_cities+next_city], d_BETA);
			d_tau[index] = (double) pow (c_phero[(cityi*n_cities)+next_city], d_ALPHA);
		}

		//synchronize
		__syncthreads();

		if(dim_index == 0){
			for(int j = 0 ; j < IROULETE ; j++){
				d_sum[ant_index] += d_eta[(ant_index*IROULETE)+j] * d_tau[(ant_index*IROULETE)+j];
			}
		}

		//synchronize
		__syncthreads();

		if (cityi == next_city || vizited(ant_index, next_city, routes, n_cities, i)) {
			d_probs[index] = 0;
		}else{
			d_probs[index] = d_eta[index] * d_tau[index] / d_sum[ant_index];
		}

		//choose next city
		if(dim_index == 0){
			if(d_sum[ant_index] > 0.0){
				int nextCity = city(ant_index, n_cities, d_probs, rand_states);
				routes[(ant_index * n_cities) + (i + 1)] = iroulette[cityi*IROULETE+nextCity];
			}else{
				int nc;
				for(nc = 0; nc < n_cities; nc++){
					if(!vizited(ant_index, nc, routes, n_cities, i)){
						break;
					}
				}
				routes[(ant_index * n_cities) + (i + 1)] = nc;
			}

			//clean for next iteration
			d_sum[ant_index] = 0.0;
		}

		d_eta[index] = 0.0;
		d_tau[index] = 0.0;

		//synchronize
		__syncthreads();
	}
}

__global__ void update_pheromones_kernel(int* NUMBEROFANTS, int* NUMBEROFCITIES, int* ROUTES, double* c_phero, double* DELTAPHEROMONES, double* DIST, double* routes_distance, double* bestRoute, int* d_best_sequence) {

//	printf("\n\n\n updatePHEROMONES: ");

	int Q = 11340;
	double RO = 0.5;

	for (int k=0; k<NUMBEROFANTS[0]; k++) {

//		printf("\n N CIties : %i", NUMBEROFCITIES[0]);

		double rlength = d_length(k, NUMBEROFCITIES[0], ROUTES, DIST);
		routes_distance[k] = rlength;

//		printf("\n Distances : %f", rlength);

		for (int r=0; r < NUMBEROFCITIES[0]-1; r++) {

			int cityi = ROUTES[k * NUMBEROFCITIES[0] + r];
			int cityj = ROUTES[k * NUMBEROFCITIES[0] + r + 1];

			DELTAPHEROMONES[cityi* NUMBEROFCITIES[0] + cityj] += Q / rlength;
			DELTAPHEROMONES[cityj* NUMBEROFCITIES[0] + cityi] += Q / rlength;
		}

		if(routes_distance[k] < bestRoute[0]){
			bestRoute[0] = routes_distance[k];
			for (int count=0; count < NUMBEROFCITIES[0]; count++) {
				d_best_sequence[count] = ROUTES[k * NUMBEROFCITIES[0]+count];
			}
		}
	}

	for (int i=0; i<NUMBEROFCITIES[0]; i++) {
		for (int j=0; j<NUMBEROFCITIES[0]; j++) {
			c_phero[i * NUMBEROFCITIES[0] + j] = (1 - RO) * c_phero[i * NUMBEROFCITIES[0] +j] + DELTAPHEROMONES[i * NUMBEROFCITIES[0] +j];
			DELTAPHEROMONES[i * NUMBEROFCITIES[0] +j] = 0.0;

			c_phero[j * NUMBEROFCITIES[0] + i] = (1 - RO) * c_phero[j * NUMBEROFCITIES[0] +i] + DELTAPHEROMONES[j * NUMBEROFCITIES[0] +i];
			DELTAPHEROMONES[j * NUMBEROFCITIES[0] +i] = 0.0;
		}
	}

	__syncthreads();
}

__device__ bool vizited(int antk, int c, int* ROUTES, int NUMBEROFCITIES, int step) {

	for (int l=0; l <= step; l++) {
		if (ROUTES[antk*NUMBEROFCITIES+l] == c) {
			return true;
		}
	}
	return false;
}

__device__ double PHI (int cityi, int cityj, int NUMBEROFCITIES, double* c_dist, double* c_phero, double sum) {

	double dista = c_dist[cityi*NUMBEROFCITIES+cityj];

	double ETAij = (double) pow (1 / dista , d_BETA);
	double TAUij = (double) pow (c_phero[(cityi * NUMBEROFCITIES) + cityj],   d_ALPHA);

	return (ETAij * TAUij) / sum;
}

__device__ int city(int antK, int NCITIES, double* PROBS, hiprandState* rand_states) {

    double random = hiprand_uniform(&rand_states[antK]);

	int i = 0;

	double sum = PROBS[antK*IROULETE];
	while (sum < random){
		i++;
		sum += PROBS[antK*IROULETE+i];
	}

	return (int) i;
}

__device__ double d_length (int antk, int NUMBEROFCITIES, int* ROUTES, double* DIST) {

	double sum = 0.0;

	for (int j=0; j<NUMBEROFCITIES-1; j++) {

		int cityi = ROUTES[antk*NUMBEROFCITIES+j];
		int cityj = ROUTES[antk*NUMBEROFCITIES+j+1];

		sum += DIST[cityi*NUMBEROFCITIES + cityj];
	}

	int cityi = ROUTES[antk*NUMBEROFCITIES+NUMBEROFCITIES-1];
	int cityj = ROUTES[antk*NUMBEROFCITIES];

	sum += DIST[cityi*NUMBEROFCITIES + cityj];

	return sum;
}

double run_aco(int n_ant, int n_iterations, int problem){

	int n_cities = 0;
	int n_ants = n_ant;

	NBLOCKS = n_ants;

	switch (problem) {
			case 1:
				n_cities = 38; //Djbouti
				break;
			case 2:
				n_cities = 980; //Luxemburg
				break;
			case 3:
				n_cities = 194; //Catar
				break;
			case 4:
				n_cities = 280;
				break;
			case 5:
				n_cities = 198;
				break;
			case 6:
				n_cities =  1291;
				break;
			case 7:
				n_cities = 318;
				break;
			case 8:
				n_cities = 442;
				break;
			case 9:
				n_cities = 1173;
				break;
			case 10:
				n_cities = 1002;
				break;
			case 11:
				n_cities = 2392;
				break;
			case 12:
				n_cities =  783;
				break;
		}

	randoms = new Randoms(15);

	int GPU_N;
	const int MAX_GPU_COUNT = 1;

	hipGetDeviceCount(&GPU_N);

	if (GPU_N > MAX_GPU_COUNT) {
		GPU_N = MAX_GPU_COUNT;
	}

	// create stream array - create one stream per GPU
	hipStream_t stream[GPU_N];

	for (int i = 0; i < GPU_N; ++i) {
		hipSetDevice(i);
		hipStreamCreate(&stream[i]);
	}

	//device variables
    double* d_coord;
    double* d_phero;
    double* d_delta_phero;
    double* d_dist;
    double* d_routes_distance;
    double* d_bestRoute;
    double* d_probs;
    double* d_eta;
    double* d_tau;
    double* d_sum;

    int* d_iroulette;
    int* d_seq;
    int* d_nants;
    int* d_ncities;
    int* d_best_sequence;


//    printf("\n Alloc vars \n");
    //Init Random Generators
    hiprandState* d_rand_states_ind;
	hipMalloc((void**)&d_rand_states_ind, n_ants * n_cities * sizeof(hiprandState));

    //alloc host variables
	double* coord = new double[n_cities*2];
	double* phero = new double[n_cities*n_cities];
	double* dist  = new double[n_cities*n_cities];
	int* best_sequence  = new int[n_cities];

//    printf("\n Alloc vars 2 \n");
    //alloc device variables
    hipMalloc((void**) &d_coord, n_cities*2*sizeof(double));
    hipMalloc((void**) &d_phero, n_cities*n_cities*sizeof(double));
    hipMalloc((void**) &d_delta_phero, n_cities*n_cities*sizeof(double));
    hipMalloc((void**) &d_dist, n_cities*n_cities*sizeof(double));
    hipMalloc((void**) &d_probs, n_ants*IROULETE*sizeof(double));
    hipMalloc((void**) &d_routes_distance, n_ants*n_cities*sizeof(double));
    hipMalloc((void**) &d_bestRoute, sizeof(double));

    hipMalloc((void**) &d_nants, sizeof(int));
    hipMalloc((void**) &d_ncities, sizeof(int));

    hipMalloc((void**) &d_best_sequence, n_cities*sizeof(int));
    hipMalloc((void**) &d_seq, n_ants*n_cities*sizeof(int));

    hipMalloc((void**) &d_sum, n_ants*sizeof(double));

    hipMalloc((void**) &d_iroulette, n_cities*IROULETE*sizeof(int));
    hipMalloc((void**) &d_eta, n_ants*IROULETE*sizeof(double));
    hipMalloc((void**) &d_tau, n_ants*IROULETE*sizeof(double));

    setup_rand_kernel<<<n_ants, n_cities, 0, stream[0]>>>(d_rand_states_ind, time(NULL));

//    printf("\n Set rand Kernel \n");

	readMap(coord,phero,dist, n_cities, problem);

//	printf("\n Read Map \n");

	double bestRoute = 99999999.9;
	hipMemcpy(d_bestRoute, &bestRoute, sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(d_phero, phero, n_cities*n_cities*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_nants, &n_ants, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_ncities, &n_cities, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_coord, coord, n_cities*2*sizeof(double), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	calculate_distance_kernel<<<1, n_cities>>>(d_dist, d_coord, n_cities); // calculates the distances of each city+
	calculate_iroulette_kernel<<<1, n_cities>>>(d_dist, d_coord, d_iroulette, n_cities); // calculates the distances of each city+

	hipDeviceSynchronize();

	hipMemcpy(dist, d_dist, (n_cities*n_cities)*sizeof(double), hipMemcpyDeviceToHost);

	//Execution Time measure
	double mean_times = 0.0;
	int iteration = 0;

	while(iteration < n_iterations){

		auto t_start = std::chrono::high_resolution_clock::now();

		route_kernel2<<<n_ants, IROULETE>>>(n_cities, d_seq, d_phero, d_dist, d_probs,d_iroulette, d_rand_states_ind, d_eta, d_tau, d_sum);
		hipDeviceSynchronize();

		auto t_end = std::chrono::high_resolution_clock::now();
		mean_times +=  std::chrono::duration<double>(t_end-t_start).count();

		update_pheromones_kernel<<<1,1>>>(d_nants, d_ncities, d_seq, d_phero, d_delta_phero, d_dist, d_routes_distance, d_bestRoute, d_best_sequence);
		hipDeviceSynchronize();

		iteration ++;
	}

	printf("\n\n Total time on Tour Construction: %f", mean_times);
	mean_times = mean_times / (n_iterations * n_ants);
	printf("\n\n Average Time on Tour Construction: %f", mean_times);

	hipDeviceSynchronize();
	hipMemcpy(best_sequence, d_best_sequence, n_cities*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&bestRoute, d_bestRoute, sizeof(double), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	printf("\n Best PATH %f \n", bestRoute);
	for (int var = 0; var < n_cities; ++var) {
		printf(" %i ", best_sequence[var]);
	}

    return bestRoute;
}
